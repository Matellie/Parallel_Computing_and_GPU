#include "hip/hip_runtime.h"
/*

This program will numerically compute the integral of

                  4/(1+x*x) 

from 0 to 1.  The value of this integral is pi -- which 
is great since it gives us an easy way to check the answer.

History: Written by Tim Mattson, 11/1999.
         Modified/extended by Jonathan Rouzaud-Cornabas, 10/2022
*/

#include <limits>
#include <cstdio>
#include <cstdlib>
#include <cstring>
//#include <sys/time.h>
#include "win-gettimeofday.h"

static long long nbSteps = 2000000000;
static int nbBlocks = 256;
static int nbThreads = 256;
double step;

__global__ void computePi(double step, long long numStep, double *result) {
  extern __shared__ double sharedSum[];

  int tid = threadIdx.x;
  long idx = blockIdx.x * blockDim.x + tid;

  double x = 0.0;
  sharedSum[tid] = 0.0;
  for (long long i = idx; i < numStep; i += blockDim.x * gridDim.x) {
    x = (i + 0.5) * step; 
    sharedSum[tid] += 4.0 / (1.0 + x * x); 
  }
  __syncthreads(); 

  for (unsigned int s = 1; s < blockDim.x ; s *= 2) {
      if (tid % (2*s) == 0) {
          sharedSum[tid] += sharedSum[tid + s];
      }
      __syncthreads(); 
  }

  if (tid == 0) {
    atomicAdd(result, sharedSum[0]);
  }
}


int main (int argc, char** argv)
{
  // Read command line arguments.
  for ( int i = 0; i < argc; i++ ) {
    if ( ( strcmp( argv[ i ], "-N" ) == 0 ) || ( strcmp( argv[ i ], "-num_steps" ) == 0 ) ) {
      nbSteps = atoll( argv[ ++i ] );
      printf( "  User num_steps is %lld\n", nbSteps );
    } else if ( ( strcmp( argv[ i ], "-h" ) == 0 ) || ( strcmp( argv[ i ], "-help" ) == 0 ) ) {
      printf( "  Pi Options:\n" );
      printf( "  -num_steps (-N) <int>:      Number of steps to compute Pi (by default 100000000)\n" );
      printf( "  -help (-h):            print this message\n\n" );
      exit( 1 );
    }
  }
      
  double pi = 0.0;

  double *cpu_sum = (double *)malloc(sizeof(double));

  step = 1.0/(double) nbSteps;

  // Timer products.
  struct timeval begin, end;

  gettimeofday( &begin, NULL );

  double *gpu_sum;
  hipError_t err = hipMalloc(&gpu_sum, sizeof(double));
  if (err != hipSuccess) {
    printf("Err:  %s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }

  computePi<<<nbBlocks, nbThreads, nbThreads * sizeof(double)>>>(step, nbSteps, gpu_sum);

  hipMemcpy(cpu_sum, gpu_sum, sizeof(double), hipMemcpyDeviceToHost);
  hipFree(gpu_sum);

  pi = step * cpu_sum[0];

  gettimeofday( &end, NULL );

  // Calculate time.
  double time = 1.0 * ( end.tv_sec - begin.tv_sec ) +
            1.0e-6 * ( end.tv_usec - begin.tv_usec );

  printf("\n pi with %lld steps is %lf in %lf seconds\n ", nbSteps, pi, time);
}
