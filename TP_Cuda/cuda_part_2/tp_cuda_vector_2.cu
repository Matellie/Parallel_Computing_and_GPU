/*
//@HEADER
// ************************************************************************
//
//                        Kokkos v. 2.0
//              Copyright (2014) Sandia Corporation
//
// Under the terms of Contract DE-AC04-94AL85000 with Sandia Corporation,
// the U.S. Government retains certain rights in this software.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are
// met:
//
// 1. Redistributions of source code must retain the above copyright
// notice, this list of conditions and the following disclaimer.
//
// 2. Redistributions in binary form must reproduce the above copyright
// notice, this list of conditions and the following disclaimer in the
// documentation and/or other materials provided with the distribution.
//
// 3. Neither the name of the Corporation nor the names of the
// contributors may be used to endorse or promote products derived from
// this software without specific prior written permission.
//
// THIS SOFTWARE IS PROVIDED BY SANDIA CORPORATION "AS IS" AND ANY
// EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
// IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
// PURPOSE ARE DISCLAIMED. IN NO EVENT SHALL SANDIA CORPORATION OR THE
// CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
// EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
// PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
// PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
// LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING
// NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS
// SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
//
// Questions Contact  H. Carter Edwards (hcedwar@sandia.gov)
//
// ************************************************************************
//@HEADER
*/


#include <hip/hip_runtime.h>
#include <limits>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <sys/time.h>
//#include "win-gettimeofday.h"

#include <cmath>

static int nbBlocks = 256;
static int nbThreads = 256;

__global__ void computeAandX(int* A, int* x, int* y, int M, int N, int* sum, int* sharedSum) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  int tid_block = threadIdx.x;
  long bid = blockIdx.x;

  sharedSum[tid] = 0;
  for (long i = tid_block; i < M; i += blockDim.x) {
    sharedSum[tid] += A[bid*M+i] * x[i]; 
  }
  __syncthreads();

  atomicAdd(sum, sharedSum[tid] * y[bid]);
}

void checkSizes( int &N, int &M,  int &S, int &nrepeat );

int main( int argc, char* argv[] ) {
  int N = -1;         // number of rows 2^12
  int M = -1;         // number of columns 2^10
  int S = -1;         // total size 2^22
  int nrepeat = 100;  // number of repeats of the test

  // Read command line arguments.
  for ( int i = 0; i < argc; i++ ) {
    if ( ( strcmp( argv[ i ], "-N" ) == 0 ) || ( strcmp( argv[ i ], "-Rows" ) == 0 ) ) {
      N = pow( 2, atoi( argv[ ++i ] ) );
      printf( "  User N is %d\n", N );
    }
    else if ( ( strcmp( argv[ i ], "-M" ) == 0 ) || ( strcmp( argv[ i ], "-Columns" ) == 0 ) ) {
      M = pow( 2, atof( argv[ ++i ] ) );
      printf( "  User M is %d\n", M );
    }
    else if ( ( strcmp( argv[ i ], "-S" ) == 0 ) || ( strcmp( argv[ i ], "-Size" ) == 0 ) ) {
      S = pow( 2, atof( argv[ ++i ] ) );
      printf( "  User S is %ld\n", S );
    }
    else if ( strcmp( argv[ i ], "-nrepeat" ) == 0 ) {
      nrepeat = atoi( argv[ ++i ] );
    }
    else if ( ( strcmp( argv[ i ], "-h" ) == 0 ) || ( strcmp( argv[ i ], "-help" ) == 0 ) ) {
      printf( "  y^T*A*x Options:\n" );
      printf( "  -Rows (-N) <int>:      exponent num, determines number of rows 2^num (default: 2^12 = 4096)\n" );
      printf( "  -Columns (-M) <int>:   exponent num, determines number of columns 2^num (default: 2^10 = 1024)\n" );
      printf( "  -Size (-S) <int>:      exponent num, determines total matrix size 2^num (default: 2^22 = 4096*1024 )\n" );
      printf( "  -nrepeat <int>:        number of repetitions (default: 100)\n" );
      printf( "  -help (-h):            print this message\n\n" );
      exit( 1 );
    }
  }

  // Check sizes.
  checkSizes( N, M, S, nrepeat );

  // Allocate x,y,A
  auto y = new int[N];
  auto x = new int[M];
  auto A = new int[N*M];


  // Initialize y vector to 1.
  for (int i = 0; i<N; i++) {
    y[i] = 1;
  }

  // Initialize x vector to 1.
  for (int i = 0; i<M; i++) {
    x[i] = 1;
  }

  // Initialize A matrix, you can use a 1D index if you want a flat structure (i.e. a 1D array) e.g. j*M+i is the same than [j][i]
  for(int i = 0; i<N; i++) {
    for(int j = 0; j<M; j++) {
      A[i*M+j] = 1;
    }
  }

  // Timer products.
  struct timeval begin, end;

  gettimeofday( &begin, NULL );

  int result = 0;
  nbBlocks = N;
  nbThreads = 256;

  int *cpu_sum = (int *)malloc(sizeof(int));

  int *gpu_sum;
  hipError_t err = hipMalloc(&gpu_sum, sizeof(int));
  if (err != hipSuccess) {
    printf("Err:  %s in %s at line %d\n", hipGetErrorString(err), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
  int *gpu_A;
  hipError_t err1 = hipMalloc(&gpu_A, N*M*sizeof(int));
  if (err1 != hipSuccess) {
    printf("Err:  %s in %s at line %d\n", hipGetErrorString(err1), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
  int *gpu_X;
  hipError_t err2 = hipMalloc(&gpu_X, M*sizeof(int));
  if (err2 != hipSuccess) {
    printf("Err:  %s in %s at line %d\n", hipGetErrorString(err2), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
  int *gpu_Y;
  hipError_t err3 = hipMalloc(&gpu_Y, N*sizeof(int));
  if (err3 != hipSuccess) {
    printf("Err:  %s in %s at line %d\n", hipGetErrorString(err3), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
  int *shared_sum;
  hipError_t err4 = hipMalloc(&shared_sum, nbBlocks * nbThreads*sizeof(int));
  if (err3 != hipSuccess) {
    printf("Err:  %s in %s at line %d\n", hipGetErrorString(err3), __FILE__, __LINE__);
    exit(EXIT_FAILURE);
  }
  hipMemcpy(gpu_A, A, N * M * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(gpu_X, x, M * sizeof(int), hipMemcpyHostToDevice);
  hipMemcpy(gpu_Y, y, N * sizeof(int), hipMemcpyHostToDevice);

  for(int repeat = 0; repeat<nrepeat; repeat++) {
    result = 0;

    computeAandX<<<nbBlocks, nbThreads>>>(gpu_A, gpu_X, gpu_Y, M, N, gpu_sum, shared_sum);
    hipMemcpy(cpu_sum, gpu_sum, sizeof(int), hipMemcpyDeviceToHost);
    result = cpu_sum[0];

    hipMemset(gpu_sum, 0, sizeof(int));

    // Output result.
    if ( repeat == ( nrepeat - 1 ) ) {
      printf( "  Computed result for %d x %d is %d\n", N, M, result );
    }

    const double solution = (double) N * (double) M;

    if ( result != S ) {
      printf( "  Error: result( %d ) != solution( %d )\n", result, S );
    }
  }


  gettimeofday( &end, NULL );

  // Calculate time.
  //double time = timer.seconds();
  double time = 1.0 * ( end.tv_sec - begin.tv_sec ) +
                1.0e-6 * ( end.tv_usec - begin.tv_usec );

  // Calculate bandwidth.
  // Each matrix A row (each of length M) is read once.
  // The x vector (of length M) is read N times.
  // The y vector (of length N) is read once.
  // double Gbytes = 1.0e-9 * double( sizeof(double) * ( 2 * M * N + N ) );
  double Gbytes = 1.0e-9 * double( sizeof(double) * ( M + M * N + N ) );

  // Print results (problem size, time and bandwidth in GB/s).
  printf( "  N( %d ) M( %d ) nrepeat ( %d ) problem( %g MB ) time( %g s ) bandwidth( %g GB/s )\n",
          N, M, nrepeat, Gbytes * 1000, time, Gbytes * nrepeat / time );

  std::free(A);
  std::free(y);
  std::free(x);

  return 0;
}

void checkSizes(  int &N,  int &M,  int &S, int &nrepeat ) {
  // If S is undefined and N or M is undefined, set S to 2^22 or the bigger of N and M.
  if ( S == -1 && ( N == -1 || M == -1 ) ) {
    S = pow( 2, 22 );
    if ( S < N ) S = N;
    if ( S < M ) S = M;
  }

  // If S is undefined and both N and M are defined, set S = N * M.
  if ( S == -1 ) S = N * M;

  // If both N and M are undefined, fix row length to the smaller of S and 2^10 = 1024.
  if ( N == -1 && M == -1 ) {
    if ( S > 1024 ) {
      M = 1024;
    } else {
      M = S;
    }
  }

  // If only M is undefined, set it.
  if ( M == -1 ) M = S / N;

  // If N is undefined, set it.
  if ( N == -1 ) N = S / M;

  printf( "  Total size S = %ld N = %ld M = %ld\n", S, N, M );

  // Check sizes.
  if ( ( S < 0 ) || ( N < 0 ) || ( M < 0 ) || ( nrepeat < 0 ) ) {
    printf( "  Sizes must be greater than 0.\n" );
    exit( 1 );
  }

  if ( ( N * M ) != S ) {
    printf( "  N * M != S\n" );
    exit( 1 );
  }
}
